
#include <hip/hip_runtime.h>
#include <cstdio>
__global__ void hello_kernel() {
    printf("Hello GPU!\n");
}

int main() {
    hello_kernel<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}
