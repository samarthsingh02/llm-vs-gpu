
#include <hip/hip_runtime.h>
// kernels/02_naive_matmul.cu
extern "C" __global__
void naive_matmul(const float* A, const float* B, float* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < N; k++) {
            sum += A[row*N + k] * B[k*N + col];
        }
        C[row*N + col] = sum;
    }
}
