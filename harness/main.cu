#include <cstdio>
#include <iostream>
#include <vector>
#include <string>
#include <functional> // Required for std::function
#include <hip/hip_runtime.h>

// Declare kernel functions
extern "C" __global__ void saxpy(float a, const float* x, float* y, int n);
extern "C" __global__ void naive_matmul(const float* A, const float* B, float* C, int N);
extern "C" __global__ void naive_transpose(const float* in, float* out, int N);
extern "C" __global__ void tiled_transpose(const float* in, float* out, int N);
extern "C" __global__ void bank_conflict(int *data);
extern "C" __global__ void strided_global(const float* in, float* out, int N, int stride);
extern "C" __global__ void branch_divergence(float* data, int N);
extern "C" __global__ void atomic_histogram(int* hist, const int* data, int N, int bins);
extern "C" __global__ void high_reg_pressure(float* data, int N);
extern "C" __global__ void parallel_reduction(float *data, float *result, int N);
// --- ADDED DECLARATION FOR NEW KERNEL ---
extern "C" __global__ void tiled_matmul(const float* A, const float* B, float* C, int N);
// --- END ADDED DECLARATION ---

// Helper to check for CUDA errors
#define CUDA_CHECK(err) { \
    hipError_t err_ = (err); \
    if (err_ != hipSuccess) { \
        std::cerr << "CUDA error: " << hipGetErrorString(err_) << " in " << __FILE__ << " at line " << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}

// Timer function
void time_kernel(const std::string& kernel_name, std::function<void()> kernel_launch, int iterations = 100) {
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    // Warm-up
    kernel_launch();
    CUDA_CHECK(hipDeviceSynchronize());

    std::cout << "Benchmarking: " << kernel_name << std::endl;

    CUDA_CHECK(hipEventRecord(start));
    for (int i = 0; i < iterations; ++i) {
        kernel_launch();
    }
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));

    float milliseconds = 0;
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));

    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    // This printf helps Nsight Compute to distinguish kernel calls
    printf("Finished benchmarking %s. Avg time: %f ms\n", kernel_name.c_str(), milliseconds / iterations);
}

int main() {
    int N = 2048; // Matrix/vector dimension
    int data_size_float = N * N * sizeof(float);
    int data_size_int = N * N * sizeof(int);

    // Allocate unified memory for simplicity
    float *dA, *dB, *dC;
    int *dIntA, *dIntB;
    CUDA_CHECK(hipMallocManaged(&dA, data_size_float));
    CUDA_CHECK(hipMallocManaged(&dB, data_size_float));
    CUDA_CHECK(hipMallocManaged(&dC, data_size_float));
    CUDA_CHECK(hipMallocManaged(&dIntA, data_size_int));
    CUDA_CHECK(hipMallocManaged(&dIntB, data_size_int));

    printf("Running kernels...\n\n");

    // 1. SAXPY
    time_kernel("saxpy", [&]() {
        saxpy<<< (N*N + 255) / 256, 256 >>>(2.0f, dA, dB, N * N);
    });

    // 2. Naive Matrix Multiply
    dim3 threadsPerBlockMatMul(16, 16); // Use a distinct variable name
    dim3 numBlocksMatMul( (N + 15) / 16, (N + 15) / 16); // Use a distinct variable name
    time_kernel("naive_matmul", [&]() {
        naive_matmul<<<numBlocksMatMul, threadsPerBlockMatMul>>>(dA, dB, dC, N);
    });

    // 3. Naive Transpose
    time_kernel("naive_transpose", [&]() {
        naive_transpose<<<numBlocksMatMul, threadsPerBlockMatMul>>>(dA, dB, N); // Can reuse grid/block dims
    });

    // 4. Tiled Transpose
    time_kernel("tiled_transpose", [&]() {
        tiled_transpose<<<numBlocksMatMul, threadsPerBlockMatMul>>>(dA, dB, N); // Can reuse grid/block dims
    });

    // 5. Shared Memory Bank Conflict
    time_kernel("bank_conflict", [&]() {
       bank_conflict<<<1, 1024>>>(dIntA);
    });

    // 6. Strided Global Access
    time_kernel("strided_global", [&]() {
        strided_global<<<(N*N + 255) / 256, 256>>>(dA, dB, N*N, 16);
    });

    // 7. Branch Divergence
    time_kernel("branch_divergence", [&]() {
        branch_divergence<<<(N*N + 255)/256, 256>>>(dA, N*N);
    });

    // 8. Atomic Contention (Histogram)
    int bins = 64;
    CUDA_CHECK(hipMemset(dIntB, 0, bins * sizeof(int))); // Initialize histogram bins
    time_kernel("atomic_histogram", [&]() {
        atomic_histogram<<<(N*N + 255)/256, 256>>>(dIntB, dIntA, N*N, bins);
    });

    // 9. High Register Pressure
    time_kernel("high_reg_pressure", [&]() {
        high_reg_pressure<<<(N*N + 255)/256, 256>>>(dA, N*N);
    });

    // 10. Parallel Reduction
    int reduction_blocks = 512;
    float* d_reduction_out;
    CUDA_CHECK(hipMallocManaged(&d_reduction_out, reduction_blocks * sizeof(float)));
    time_kernel("parallel_reduction", [&]() {
        // Shared memory size is blockDim.x * sizeof(float), blockDim.x is 512 here
        parallel_reduction<<<reduction_blocks, 512, 512 * sizeof(float)>>>(dA, d_reduction_out, N*N);
    });

    // --- ADDED CALL FOR NEW KERNEL ---
    // 11. Tiled Matrix Multiply
    // Use TILE_DIM=32 as defined in the kernel
    dim3 threadsPerBlockTiled(32, 32);
    dim3 numBlocksTiled( (N + 31) / 32, (N + 31) / 32);
    time_kernel("tiled_matmul", [&]() {
        tiled_matmul<<<numBlocksTiled, threadsPerBlockTiled>>>(dA, dB, dC, N);
    });
    // --- END ADDED CALL ---

    printf("\nAll kernels executed.\n");

    // Free memory
    CUDA_CHECK(hipFree(dA));
    CUDA_CHECK(hipFree(dB));
    CUDA_CHECK(hipFree(dC));
    CUDA_CHECK(hipFree(dIntA));
    CUDA_CHECK(hipFree(dIntB));
    CUDA_CHECK(hipFree(d_reduction_out));

    return 0;
}